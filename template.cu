#include "hip/hip_runtime.h"
/*
Copyright (C) Muaaz Gul Awan and Fahad Saeed 
This program is free software; you can redistribute it and/or
modify it under the terms of the GNU General Public License
as published by the Free Software Foundation; either version 2
of the License, or (at your option) any later version.
This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.
You should have received a copy of the GNU General Public License
along with this program; if not, write to the Free Software
Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301, USA
*/



#include<iostream>
#include<string>
#include <vector>
#include <time.h>
#include <random>
using namespace std;


__global__ void launchKernel (int *d_mySimpleData);
__device__ int simplifiedOperation(int myData);
__device__ resultSifting(int* myResult);
void PostProcessingFunctions(int *h_myResult);

int main()
{

	/***initializing sample data***
	 2-D array of floats
	 */
	float **myData = new float*[100];
	for(int i = 0; i < 100; i++){
		myData[i] = new float[100];
	}

	srand(time());

	for(int i = 0; i < 100; i++){
		for(int j = 0; j < 100; j++){
			myData[i][j] = rand()%100;
		}
	}

	/**** STEP-1 ***
	 * Simplifying floating point numbers
	 * using a user-defined threshold to
	 * convert them into zeros and ones
	 */

	int **mySimpleData = new float*[100];
	for(int i = 0; i < 100; i++){
		mySimpleData[i] = new float[100];
	}

	float threshold = 50; // can be any suitable number

	for(int i = 0; i < 100; i++){
			for(int j = 0; j < 100; j++){
				mySimpleData[i][j] = myData[i][j]>threshold?1:0;
			}
		}

	// freeing any space
	for(int i = 0; i < 100; i++){
		delete[] myData[i];
	delete[] myData;
	}

	int *d_mySimpleData; // device variable
	int *d_myResult; // variable to store result on device
	int *h_myResult = new int[]; // suitable sized array for result
	hipMalloc((void**) &d_mySimpleData, 100*100*sizeof(int)); // Assigning memory on device
	hipMalloc((void**) &d_myResult, 100*100*sizeof(int)); // Assigning memory on device
	hipMemcpy(d_mySimpleData, mySimpleData, sizeof(int)*100*100, hipMemcpyHostToDevice); // copying to device via PCIe
	launchKernel<<<100,100>>>(d_mySimpleData); // launch kernel function on device
	hipMemcpy(h_myResult, 	hipMemcpy(d_mySimpleData, mySimpleData, sizeof(int)*100*100, hipMemcpyHostToDevice); //Copy back results via PCIe

	PostProcessingFunctions(h_myResult);

}

/*Step-3
 * fine grained mapping such that
 * each data point is assigned to
 * a unique compute unit, for
 * example of 2-step mapping refer to
 * GPU-ArraySort code
 */

__global__ void launchKernel (int *d_mySimpleData, int *myResult){
	int myID = threadIdx.x + BlockIdx.x*blockDim.x; // for fine grained mappping (STEP-3).

	__shared__ int myShMem[100*100];

	//moving to shared memory (STEP-4)
	myShMem[myID] = d_mySimpleData[myID];

	myResult = simplifiedOperation(myShMem);

	resultSifting (myResult); // in case result is too large, suitable result sifting operation needs to be performed.


}

/*
 * STEP-2
 * simplified operations to be performed on
 * simplified data. User can replace this with
 * desired F_sub.
 */
__device__ int simplifiedOperation(int myData){
	// perform operation on myData in-place.
}
/* STEP-6
 * result sifting operation to filter out interesting
 * results or to compress the large result DS into
 * compact ones. For more example refer to G-MSR code
 */
__device__ resultSifting(int* myResult){
	//filter out result using
	//suitable functions
}

/*
 * STEP-7
 * post processing phase to complete
 * the processing and bring the GPU
 * results in more suitable form
 */
void PostProcessingFunctions(int *h_myResult){
	//perform suitable post processing operations on CPU side
	//for a detailed example refer to G-MSR code where binary
	//spectra are transformed to reduced spectra in post
	//processing phase
}
